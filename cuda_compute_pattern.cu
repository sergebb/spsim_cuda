#include "hip/hip_runtime.h"
/*
 * File:   cuda_compute_pattern.cu
 * Author: Sergey Bobkov
 *
 * Created on October 28, 2014, 12:35 PM
 */

#include <hip/hip_runtime_api.h>
#include "cuda_compute_pattern.h"
#define BLOCK_DIM 256

#define CUDA_CHECK_ERROR(err)           \
	if (err != hipSuccess) {          \
	printf("Cuda error: %s\n", hipGetErrorString(err));    \
	printf("Error in file: %s, line: %i\n", __FILE__, __LINE__);  \
	}


texture<float, 2, hipReadModeElementType> tex_atoms;

texture<int, hipTextureType1D, hipReadModeElementType> tex_atomic_number;

__device__ float scatt_factor(float d,int Z){
  int i;
  /* the 0.25 is there because the formula is (|H|/2)^2 */
  float res = tex2D(tex_atoms,8,Z);
  for(i = 0;i<4;i++){
    res+= tex2D(tex_atoms,i,Z)*exp((-tex2D(tex_atoms,i+4,Z))*d*d*0.25*1e-20);
  }
  return res;
}

__global__ void cuda_kernel_compute_pattern( int HKL_list_size, int natoms, float *dev_HKL_list, float * dev_pos, float *dev_i, int *dev_is_element_in_molecule) {
    int tNum = blockIdx.x * blockDim.x + threadIdx.x;
    double scattering_factor;
    double scattering_vector_length;
    double scattering_factor_cache[ELEMENTS];
    double dev_F_R,dev_F_I;

    if (tNum < HKL_list_size) {
        dev_F_R = 0;
        dev_F_I = 0;
        float* pixel_coord = dev_HKL_list+3*tNum;
        scattering_vector_length = sqrt(pixel_coord[0]*pixel_coord[0] + pixel_coord[1]*pixel_coord[1] + pixel_coord[2]*pixel_coord[2]);
//        scattering_vector_length = sqrt(dev_HLK_list[tNum]*dev_HLK_list[tNum] + dev_HLK_list[tNum + HKL_list_size]*dev_HLK_list[tNum + HKL_list_size] + dev_HLK_list[tNum + 2*HKL_list_size]*dev_HLK_list[tNum + 2*HKL_list_size]);
        for (int j = 0; j < ELEMENTS; j++) {
           if (dev_is_element_in_molecule[j]) {
                scattering_factor_cache[j] = scatt_factor(scattering_vector_length, j);
            }
        }
        for (int j = 0; j < natoms; j++) {
            scattering_factor = scattering_factor_cache[ tex1Dfetch(tex_atomic_number,j)];
            double sin_phase, cos_phase;
            sincos( 2 * M_PI * (pixel_coord[0] * dev_pos[3*j] + pixel_coord[1]* dev_pos[3*j + 1] + pixel_coord[2] * dev_pos[3*j + 2]), &sin_phase, &cos_phase );
            dev_F_R += scattering_factor * cos_phase;
            dev_F_I += scattering_factor * sin_phase;
        }
        dev_i[tNum] = (float)(dev_F_R * dev_F_R + dev_F_I * dev_F_I);
    }
}

//__global__ void cuda_kernel_mark_molecule( int natoms, int * dev_atomic_number, int *dev_is_element_in_molecule) {
//    int threadNum = blockIdx.x * blockDim.x + threadIdx.x;
//
//    if (threadNum < ELEMENTS) {
//        dev_is_element_in_molecule[threadNum] = 0;
//    }
//    __syncthreads();
//    if (threadNum < natoms) {
//        dev_is_element_in_molecule[dev_atomic_number[threadNum]] = 1;
//    }
//}

__global__ void cuda_kernel_init( int HKL_list_size, int natoms, int * dev_atomic_number, int *dev_is_element_in_molecule, double *dev_f, float *dev_i) {
    int threadNum = blockIdx.x * blockDim.x + threadIdx.x;

    if (threadNum < ELEMENTS) {
        dev_is_element_in_molecule[threadNum] = 0;
    }
    __syncthreads();

    if (threadNum < natoms ) {
        dev_is_element_in_molecule[dev_atomic_number[threadNum]] = 1;
    }
}

__global__ void cuda_kernel_solid_angle( float *det_solid_angle, int det_nx, int det_ny, float det_width,
        float det_height, float det_pixel_width, float det_pixel_height, float det_distance){

    int tNum = blockIdx.x * blockDim.x + threadIdx.x;
    double px,py;
    double corners[4][2];
    double corner_distance[4];
    double projected_pixel_sides[4];
    double r;
    int x = tNum / det_ny;
    int y = tNum % det_ny;
    int i;
    if (tNum < det_nx * det_ny) {
        px = ((x - (det_nx - 1.0) / 2.0) / det_nx) * det_width / 2;
        py = (((det_ny - 1.0) / 2.0 - y) / det_ny) * det_height / 2;
        r = sqrt(det_distance * det_distance + px * px + py * py);
        /* top left */
        corners[0][0] = px - det_pixel_width / 2;
        corners[0][1] = py + det_pixel_height / 2;
        corner_distance[0] = sqrt(det_distance * det_distance + corners[0][0] * corners[0][0] + corners[0][1] * corners[0][1]);
        /* top right */
        corners[1][0] = px + det_pixel_width / 2;
        corners[1][1] = py + det_pixel_height / 2;
        corner_distance[1] = sqrt(det_distance * det_distance + corners[1][0] * corners[1][0] + corners[1][1] * corners[1][1]);
        /* bottom right */
        corners[2][0] = px + det_pixel_width / 2;
        corners[2][1] = py - det_pixel_height / 2;
        corner_distance[2] = sqrt(det_distance * det_distance + corners[2][0] * corners[2][0] + corners[2][1] * corners[2][1]);
        /* bottom left */
        corners[3][0] = px - det_pixel_width / 2;
        corners[3][1] = py - det_pixel_height / 2;
        corner_distance[3] = sqrt(det_distance * det_distance + corners[3][0] * corners[3][0] + corners[3][1] * corners[3][1]);
        /* project on plane*/
        for (i = 0; i < 4; i++) {
            corners[i][0] *= r / corner_distance[i];
            corners[i][1] *= r / corner_distance[i];
        }
        /* top */
        projected_pixel_sides[0] = sqrt((corners[0][0] - corners[1][0])*(corners[0][0] - corners[1][0])+(corners[0][1] - corners[1][1])*(corners[0][1] - corners[1][1]));
        /* left */
        projected_pixel_sides[1] = sqrt((corners[0][0] - corners[3][0])*(corners[0][0] - corners[3][0])+(corners[0][1] - corners[3][1])*(corners[0][1] - corners[3][1]));
        det_solid_angle[tNum] = projected_pixel_sides[0] * projected_pixel_sides[1] / (r * r);

    }
}

/*
 *
 */
float *cuda_compute_pattern_on_list(Molecule * mol, float * HKL_list, int HKL_list_size,float atomsf[][9]){
    float *dev_HLK_list;
    int * dev_atomic_number;
    float * dev_pos;
    float *res_intencity;
//    double *res_f;
    float *dev_i;
    double *dev_f;
    hipArray *dev_atomsf;

    hipEvent_t start, stop;
    float time;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);


//    float *tmp_HKL_list = new float[3*HKL_list_size];
//    for(int i=0;i<HKL_list_size;i++){
//        tmp_HKL_list[i] = HKL_list[3*i];
//        tmp_HKL_list[i+HKL_list_size] = HKL_list[3*i+1];
//        tmp_HKL_list[i+2*HKL_list_size] = HKL_list[3*i+2];
//    }
//    CUDA_CHECK_ERROR(hipMemcpy(dev_HLK_list, tmp_HKL_list, 3*sizeof (float)*HKL_list_size, hipMemcpyHostToDevice));
//    delete[] tmp_HKL_list;

//    float *tmp_pos = new float[3*mol->natoms];
//    for(int i=0;i<mol->natoms;i++){
//        tmp_pos[i] = mol->pos[3*i];
//        tmp_pos[i+mol->natoms] = mol->pos[3*i+1];
//        tmp_pos[i+2*mol->natoms] = mol->pos[3*i+2];
//    }
//    CUDA_CHECK_ERROR(hipMemcpy(dev_pos, tmp_pos, 3*sizeof (float) * mol->natoms, hipMemcpyHostToDevice));
//    delete[] tmp_pos;

    CUDA_CHECK_ERROR(hipMalloc((void**) &(dev_HLK_list), 3*sizeof (float) * HKL_list_size));
    CUDA_CHECK_ERROR(hipMemcpy(dev_HLK_list, HKL_list, 3*sizeof (float)*HKL_list_size, hipMemcpyHostToDevice));

    CUDA_CHECK_ERROR(hipMalloc((void**) &(dev_atomic_number), sizeof (int) * mol->natoms));
    CUDA_CHECK_ERROR(hipMemcpy(dev_atomic_number, mol->atomic_number, sizeof (int) * mol->natoms, hipMemcpyHostToDevice));
    CUDA_CHECK_ERROR(hipBindTexture((size_t)0,tex_atomic_number, dev_atomic_number, sizeof (int) * mol->natoms));

    CUDA_CHECK_ERROR(hipMalloc((void**) &(dev_pos), 3*sizeof (float) * mol->natoms));
    CUDA_CHECK_ERROR(hipMemcpy(dev_pos, mol->pos, 3*sizeof (float) * mol->natoms, hipMemcpyHostToDevice));

    CUDA_CHECK_ERROR(hipMalloc((void**) &(dev_f), 2*sizeof(double)*HKL_list_size));
    CUDA_CHECK_ERROR(hipMalloc((void**) &(dev_i), sizeof(float)*HKL_list_size));
//    CUDA_CHECK_ERROR(hipHostMalloc((void**) &(res_intencity), sizeof(float)*HKL_list_size));
//    CUDA_CHECK_ERROR(hipHostMalloc((void**) &(res_f), 2*sizeof(double)*HKL_list_size));
    res_intencity = (float*)malloc(sizeof(float)*HKL_list_size);

    int *dev_is_element_in_molecule;
    CUDA_CHECK_ERROR(hipMalloc((void**) &(dev_is_element_in_molecule), sizeof(int)*ELEMENTS));

    //Create Channel Descriptor. float is just for example. Change it to required data type.
    hipChannelFormatDesc channel = hipCreateChannelDesc<float>();

    //Allocate Memory
    CUDA_CHECK_ERROR(hipMallocArray(&dev_atomsf, &channel, 9, ELEMENTS, hipArrayDefault));
    CUDA_CHECK_ERROR(hipMemcpyToArray(dev_atomsf, 0, 0, atomsf, sizeof(float)*9*ELEMENTS, hipMemcpyHostToDevice));
    CUDA_CHECK_ERROR(hipBindTextureToArray(tex_atoms, dev_atomsf, channel));

    dim3 gridSize = dim3(HKL_list_size % BLOCK_DIM == 0 ? HKL_list_size / BLOCK_DIM : HKL_list_size / BLOCK_DIM + 1, 1, 1);
    dim3 gridSizeAtoms = dim3(ELEMENTS % BLOCK_DIM == 0 ? ELEMENTS / BLOCK_DIM : ELEMENTS / BLOCK_DIM + 1, 1, 1);
    dim3 blockSize = dim3(BLOCK_DIM, 1, 1);

    cuda_kernel_init<<< gridSize, blockSize >>>( HKL_list_size, mol->natoms, dev_atomic_number, dev_is_element_in_molecule, dev_f, dev_i);
    cuda_kernel_compute_pattern<<< gridSize, blockSize >>>(HKL_list_size, mol->natoms, dev_HLK_list, dev_pos, dev_i, dev_is_element_in_molecule);

    CUDA_CHECK_ERROR(hipDeviceSynchronize());

    CUDA_CHECK_ERROR(hipMemcpy(res_intencity, dev_i, sizeof(float)*HKL_list_size, hipMemcpyDeviceToHost));
//    CUDA_CHECK_ERROR(hipMemcpy(res_f, dev_f, 2*sizeof(double)*HKL_list_size, hipMemcpyDeviceToHost));

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
//    printf ("Time for the gpu compute: %f ms\n", time);

    CUDA_CHECK_ERROR(hipFreeArray(dev_atomsf));

    CUDA_CHECK_ERROR(hipFree(dev_HLK_list));
    CUDA_CHECK_ERROR(hipFree(dev_atomic_number));
    CUDA_CHECK_ERROR(hipFree(dev_pos));
    CUDA_CHECK_ERROR(hipFree(dev_f));
    CUDA_CHECK_ERROR(hipFree(dev_i));
    CUDA_CHECK_ERROR(hipFree(dev_is_element_in_molecule));


    return res_intencity;

}


float *cuda_calculate_pixel_solid_angle( int det_nx, int det_ny, float det_width,
        float det_height, float det_pixel_width, float det_pixel_height, float det_distance){

    float *result;
    float *dev_solid_angle;

    CUDA_CHECK_ERROR(hipMalloc((void**) &(dev_solid_angle), sizeof(float)*det_nx*det_ny));
    CUDA_CHECK_ERROR(hipHostMalloc((void**) &(result), sizeof(float)*det_nx*det_ny));
    result = (float*)malloc(sizeof(float)*det_nx*det_ny);

    dim3 gridSize = dim3(det_nx*det_ny % BLOCK_DIM == 0 ? det_nx*det_ny / BLOCK_DIM : det_nx*det_ny / BLOCK_DIM + 1, 1, 1);
    dim3 blockSize = dim3(BLOCK_DIM, 1, 1);

    cuda_kernel_solid_angle<<< gridSize, blockSize >>>(dev_solid_angle, det_nx, det_ny, det_width,
        det_height, det_pixel_width, det_pixel_height, det_distance);

    CUDA_CHECK_ERROR(hipDeviceSynchronize());
    CUDA_CHECK_ERROR(hipMemcpy(result, dev_solid_angle, sizeof(float)*det_nx*det_ny, hipMemcpyDeviceToHost));

    CUDA_CHECK_ERROR(hipFree(dev_solid_angle));

    return result;


}
